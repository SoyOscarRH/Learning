#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void sum_matrix_kernel(const int* matrix_1_device, const int* matrix_2_device,
                                  int* result_device, int size) {
  const int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) result_device[i] = matrix_1_device[i] + matrix_2_device[i];
}

void sum_matrix_at_cuda(const int* matrix_1_host, const int* matrix_2_host, int* result_host,
                        const int size) {
  int *result_device = NULL, *matrix_1_device = NULL, *matrix_2_device = NULL;
  const int how_many = size * sizeof(int);

  hipMalloc((void**)&result_device, how_many);
  hipMalloc((void**)&matrix_1_device, how_many);
  hipMalloc((void**)&matrix_2_device, how_many);

  hipMemcpy(matrix_1_device, matrix_1_host, how_many, hipMemcpyHostToDevice);
  hipMemcpy(matrix_2_device, matrix_2_host, how_many, hipMemcpyHostToDevice);

  sum_matrix_kernel<<<2, (how_many + 1) / 2>>>(matrix_1_device, matrix_2_device, result_device,
                                               how_many);

  hipDeviceSynchronize();

  hipMemcpy(result_host, result_device, how_many, hipMemcpyDeviceToHost);

  hipFree(result_device);
  hipFree(matrix_1_device);
  hipFree(matrix_2_device);
  hipDeviceReset();
}

void print_matrix(const int* matrix, int x, int y) {
  printf("{");
  for (int i = 0; i < x; ++i) {
    printf("{");
    for (int j = 0; j < y; ++j) printf("%i, ", matrix[i * x + j]);
    printf("}");
  }
  printf("}");
}
