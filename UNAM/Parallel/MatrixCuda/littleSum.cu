#include "hip/hip_runtime.h"
#include <time.h>
#include <math.h>

#include "sum.cu"

int main() {
  const int x = 3, y = 3;
  const int size = x * y;

  const int intput_1[x][y] = {{1, 2, 3}, {1, 2, 3}, {1, 2, 3}};
  const int intput_2[x][y] = {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}};

  int result[x][y] = {0};

  clock_t start = clock();
  sum_matrix_at_cuda((int*)intput_1, (int*)intput_2, (int*)result, size);
  clock_t end = clock();

  double clocks = fabs(end - start);
  double time_taken = clocks / CLOCKS_PER_SEC;
  printf("Time taken: %f\n", time_taken);

  print_matrix((int*)intput_1, x, y);
  printf(" + ");
  print_matrix((int*)intput_2, x, y);
  printf(" = ");
  print_matrix((int*)result, x, y);
  printf("\n");

  return 0;
}
