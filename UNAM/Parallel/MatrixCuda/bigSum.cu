#include "hip/hip_runtime.h"
#include <time.h>

#include "sum.cu"

int main() {
  srand(time(0));

  const int x = 300, y = 300;
  const int size = x * y;

  int* intput_1 = (int*)malloc(size * sizeof(int));
  int* intput_2 = (int*)malloc(size * sizeof(int));
  int result[x][y] = {0};

  for (int i = 0; i < size; ++i) intput_1[i] = rand() % 5, intput_2[i] = rand() % 5;

  clock_t start = clock();
  sum_matrix_at_cuda((int*)intput_1, (int*)intput_2, (int*)result, size);
  clock_t end = clock();

  double clocks = fabs(end - start);
  double time_taken = clocks / CLOCKS_PER_SEC;
  printf("Time taken: %f\n", time_taken);

  // print_matrix((int*)intput_1, x, y);
  // printf(" + ");
  // print_matrix((int*)intput_2, x, y);
  // printf(" = ");
  // print_matrix((int*)result, x, y);
  // printf("\n");

  return 0;
}
