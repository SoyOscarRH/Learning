#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

__global__ void matrix_multiplication(int* m, int* n, int* p, int size) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int p_sum;

  for (int i = 0; i < size; i++) {
    p_sum += m[row * size + i] * n[col * size + i];
  }

  p[row * size + col] = p_sum;
}

int main() {
  // numero de elementos
  int n = 1024;

  // matrices a manejar
  int* h_m;
  int* h_n;
  int* h_p;

  // matrices a manejar en el device
  int* d_m;
  int* d_n;
  int* d_p;

  size_t bytes = n * n * sizeof(int);

  // alocamos memoria en el host
  h_m = (int*)malloc(bytes);
  h_n = (int*)malloc(bytes);
  h_p = (int*)malloc(bytes);

  // inicializamos las matrices

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      h_m[i * n + j] = rand() % 1024;
      h_n[i * n + j] = rand() % 1024;
    }
  }

  // alocar memoria en el device

  hipMalloc(&d_m, bytes);
  hipMalloc(&d_n, bytes);
  hipMalloc(&d_p, bytes);

  // copiando al device
  hipMemcpy(d_m, h_m, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_n, h_n, bytes, hipMemcpyHostToDevice);

  int threads_per_block = 16;
  dim3 block_size(threads_per_block, threads_per_block);  // hay 16 bloques con 16 hilos

  dim3 grid_size(n / block_size.x, n / block_size.y);

  // llamar el kernel
  matrix_multiplication<<<grid_size, block_size>>>(d_m, d_n, d_p, n);
  hipDeviceSynchronize();

  // copiar al host

  hipMemcpy(h_p, d_p, bytes, hipMemcpyDeviceToHost);

  printf("%d", h_p[0]);

  return 0;
}